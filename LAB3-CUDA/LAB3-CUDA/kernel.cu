﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <vector>
#include <string>


#include <sstream>


struct Employee {
    char name[256];
    int year;
    float salary;
};

struct Result {
    char data[256];
};

__device__ const char* salaryFromScore(float s) {
    if (s >= 50000) return "TOP";
    else if (s >= 25000) return "AVERAGE";
    else return "LEAST";
}

__device__ void toUpperString(char* str) {
    for (int i = 0; str[i] != '\0'; i++) {
        if (str[i] >= 'a' && str[i] <= 'z') {
            str[i] = str[i] - 'a' + 'A';
        }
    }
}

std::vector<Employee> read(const char* filename) {
    std::vector<Employee> employees;
    std::ifstream file(filename);
    std::string line;

    while (std::getline(file, line)) {
        Employee s;
        sscanf(line.c_str(), "%[^,],%d,%f", s.name, &s.year, &s.salary);
        employees.push_back(s);
    }

    file.close();
    return employees;
}

__global__ void process(Employee* employees, Result* results, int numEmployees, int* resultCounter) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // the starting index of the current block in the grid adding the index of the thread
    if (idx < numEmployees) {
        Employee s = employees[idx];
        if (s.name[0] > 'O') {
            toUpperString(s.name);

            char computed[256];
            int indexing = 0;
            for (int i = 0; s.name[i] != '\0'; i++) {
                computed[indexing++] = s.name[i];
            }
            computed[indexing++] = '-';
            computed[indexing++] = '0'+ s.year;
            computed[indexing++] = '-';

            const char* ss = salaryFromScore(s.salary);
            int i;
            for (i = 0; ss[i] != '\0'; i++) {
                computed[indexing++] = ss[i];
            }
            computed[indexing] = '\0';  // Add a null terminator to the end of computed

            

            // Write to result array using atomic operation
            int jjj = atomicAdd(resultCounter, 1);
            for (int i = 0; computed[i] != '\0'; i++) {
                results[jjj].data[i] = computed[i];
            }
            results[jjj].data[indexing] = '\0'; // Null-terminate the result
        }
    }
}

int main() {
    
    std::vector<Employee> employeeVector = read("Employees.txt");
    int numEmployees = employeeVector.size();

    // Prepare arrays for CUDA

    Result* h_results = new Result[numEmployees];
    int h_resultCounter = 0;

    
    // Allocate memory on GPU
    Employee* d_employees;
    Result* d_results;
    int* d_resultCounter;
    hipMalloc(&d_employees, numEmployees * sizeof(Employee));
    hipMalloc(&d_results, numEmployees * sizeof(Result));
    hipMalloc(&d_resultCounter, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_employees, employeeVector.data(), numEmployees * sizeof(Employee), hipMemcpyHostToDevice);
    hipMemcpy(d_resultCounter, &h_resultCounter, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 64; 
    int numBlocks = (numEmployees + blockSize - 1) / blockSize;
    process <<<numBlocks, blockSize >>> (d_employees, d_results, numEmployees, d_resultCounter);


    hipMemcpy(h_results, d_results, numEmployees * sizeof(Result), hipMemcpyDeviceToHost);
    hipMemcpy(&h_resultCounter, d_resultCounter, sizeof(int), hipMemcpyDeviceToHost);


    std::ofstream outFile("results.txt");
    for (int i = 0; i < h_resultCounter; i++) {
        outFile << h_results[i].data << std::endl;
    }
    outFile.close();


    hipFree(d_employees);
    hipFree(d_results);
    hipFree(d_resultCounter);

    return 0;
}